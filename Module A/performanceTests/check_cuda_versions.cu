#include <cstdio>
#include <hip/hip_runtime.h>
int main(){
  int rv = 0, dv = 0;
  hipRuntimeGetVersion(&rv);
  hipDriverGetVersion(&dv);
  printf("cudaRuntimeGetVersion: %d\ncudaDriverGetVersion: %d\n", rv, dv);
  return 0;
}
